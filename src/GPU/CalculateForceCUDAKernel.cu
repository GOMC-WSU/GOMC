
#include <hip/hip_runtime.h>
/*******************************************************************************
GPU OPTIMIZED MONTE CARLO (GOMC) 2.40
Copyright (C) 2018  GOMC Group
A copy of the GNU General Public License can be found in the COPYRIGHT.txt
along with this program, also can be found at <http://www.gnu.org/licenses/>.
********************************************************************************/
#ifdef GOMC_CUDA

#include <cuda.h>
#include "CalculateForceCUDAKernel.cuh"
#include "CalculateEnergyCUDAKernel.cuh"
#include "ConstantDefinitionsCUDAKernel.cuh"
#include "CalculateMinImageCUDAKernel.cuh"
#include "cub/cub.cuh"
#include <stdio.h>

using namespace cub;

void CallBoxInterForceGPU(VariablesCUDA *vars,
                          vector<uint> &pair1,
                          vector<uint> &pair2,
                          XYZArray const &currentCoords,
                          XYZArray const &currentCOM,
                          BoxDimensions const &boxAxes,
                          bool electrostatic,
                          vector<double> &particleCharge,
                          vector<int> &particleKind,
                          vector<int> &particleMol,
                          double &rT11,
                          double &rT12,
                          double &rT13,
                          double &rT22,
                          double &rT23,
                          double &rT33,
                          double &vT11,
                          double &vT12,
                          double &vT13,
                          double &vT22,
                          double &vT23,
                          double &vT33,
                          uint const box)
{
  int atomNumber = currentCoords.Count();
  int molNumber = currentCOM.Count();
  int *gpu_pair1, *gpu_pair2;
  int *gpu_particleKind;
  int *gpu_particleMol;
  int blocksPerGrid, threadsPerBlock;
  double *gpu_particleCharge;
  double *gpu_final_value;

  cudaMalloc((void**) &gpu_pair1, pair1.size() * sizeof(int));
  cudaMalloc((void**) &gpu_pair2, pair2.size() * sizeof(int));
  cudaMalloc((void**) &gpu_particleCharge,
             particleCharge.size() * sizeof(double));
  cudaMalloc((void**) &gpu_particleKind, particleKind.size() * sizeof(int));
  cudaMalloc((void**) &gpu_particleMol, particleMol.size() * sizeof(int));
  cudaMalloc((void**) &gpu_final_value, sizeof(double));

  cudaMemcpy(gpu_pair1, &pair1[0], pair1.size() * sizeof(int),
             cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_pair2, &pair2[0], pair2.size() * sizeof(int),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_x, currentCoords.x, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, currentCoords.y, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, currentCoords.z, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_comx, currentCOM.x, molNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_comy, currentCOM.y, molNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_comz, currentCOM.z, molNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleCharge, &particleCharge[0],
             particleCharge.size() * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleKind, &particleKind[0],
             particleKind.size() * sizeof(int),
             cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleMol, &particleMol[0],
             particleMol.size() * sizeof(int),
             cudaMemcpyHostToDevice);

  // Run the kernel...
  threadsPerBlock = 256;
  blocksPerGrid = (int)(pair1.size() / threadsPerBlock) + 1;
  BoxInterForceGPU <<< blocksPerGrid, threadsPerBlock>>>(gpu_pair1,
      gpu_pair2,
      vars->gpu_x,
      vars->gpu_y,
      vars->gpu_z,
      vars->gpu_comx,
      vars->gpu_comy,
      vars->gpu_comz,
      boxAxes.GetAxis(box).x,
      boxAxes.GetAxis(box).y,
      boxAxes.GetAxis(box).z,
      electrostatic,
      gpu_particleCharge,
      gpu_particleKind,
      gpu_particleMol,
      vars->gpu_rT11,
      vars->gpu_rT12,
      vars->gpu_rT13,
      vars->gpu_rT22,
      vars->gpu_rT23,
      vars->gpu_rT33,
      vars->gpu_vT11,
      vars->gpu_vT12,
      vars->gpu_vT13,
      vars->gpu_vT22,
      vars->gpu_vT23,
      vars->gpu_vT33,
      pair1.size(),
      vars->gpu_sigmaSq,
      vars->gpu_epsilon_Cn,
      vars->gpu_n,
      vars->gpu_VDW_Kind,
      vars->gpu_isMartini,
      vars->gpu_count,
      vars->gpu_rCut,
      vars->gpu_rCutCoulomb,
      vars->gpu_rCutLow,
      vars->gpu_rOn,
      vars->gpu_alpha,
      vars->gpu_ewald,
      vars->gpu_diElectric_1,
      vars->gpu_cell_x[box],
      vars->gpu_cell_y[box],
      vars->gpu_cell_z[box],
      vars->gpu_Invcell_x[box],
      vars->gpu_Invcell_y[box],
      vars->gpu_Invcell_z[box],
      vars->gpu_nonOrth,
      box);
  cudaDeviceSynchronize();
  // ReduceSum // Virial of LJ
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT11,
                    gpu_final_value, pair1.size());
  cudaMalloc(&d_temp_storage, temp_storage_bytes);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT11,
                    gpu_final_value, pair1.size());
  cudaMemcpy(&vT11, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT12,
                    gpu_final_value, pair1.size());
  cudaMemcpy(&vT12, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT13,
                    gpu_final_value, pair1.size());
  cudaMemcpy(&vT13, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT22,
                    gpu_final_value, pair1.size());
  cudaMemcpy(&vT22, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT23,
                    gpu_final_value, pair1.size());
  cudaMemcpy(&vT23, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT33,
                    gpu_final_value, pair1.size());
  cudaMemcpy(&vT33, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);

  if(electrostatic) {
    // ReduceSum // Virial of Coulomb
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT11,
                      gpu_final_value, pair1.size());
    cudaMemcpy(&rT11, gpu_final_value, sizeof(double),
               cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT12,
                      gpu_final_value, pair1.size());
    cudaMemcpy(&rT12, gpu_final_value, sizeof(double),
               cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT13,
                      gpu_final_value, pair1.size());
    cudaMemcpy(&rT13, gpu_final_value, sizeof(double),
               cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT22,
                      gpu_final_value, pair1.size());
    cudaMemcpy(&rT22, gpu_final_value, sizeof(double),
               cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT23,
                      gpu_final_value, pair1.size());
    cudaMemcpy(&rT23, gpu_final_value, sizeof(double),
               cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT33,
                      gpu_final_value, pair1.size());
    cudaMemcpy(&rT33, gpu_final_value, sizeof(double),
               cudaMemcpyDeviceToHost);
  }

  cudaFree(d_temp_storage);
  cudaFree(gpu_pair1);
  cudaFree(gpu_pair2);
  cudaFree(gpu_particleKind);
  cudaFree(gpu_particleMol);
  cudaFree(gpu_particleCharge);
  cudaFree(gpu_final_value);
}

void CallBoxForceGPU(VariablesCUDA *vars,
                     vector<uint> pair1,
                     vector<uint> pair2,
                     XYZArray const &coords,
                     BoxDimensions const &boxAxes,
                     bool electrostatic,
                     vector<double> particleCharge,
                     vector<int> particleKind,
                     vector<int> particleMol,
                     double &REn,
                     double &LJEn,
                     double *aForcex,
                     double *aForcey,
                     double *aForcez,
                     double *mForcex,
                     double *mForcey,
                     double *mForcez,
                     int atomCount,
                     int molCount,
                     bool reset_force,
                     bool copy_back,
                     uint const box)
{
  int atomNumber = coords.Count();
  int *gpu_pair1, *gpu_pair2, *gpu_particleKind, *gpu_particleMol;
  int blocksPerGrid, threadsPerBlock;
  double *gpu_particleCharge;
  double *gpu_REn, *gpu_LJEn;
  double *gpu_final_REn, *gpu_final_LJEn;
  double cpu_final_REn, cpu_final_LJEn;

  if(reset_force) {
    cudaMemset(vars->gpu_aForcex, 0, atomCount * sizeof(double));
    cudaMemset(vars->gpu_aForcey, 0, atomCount * sizeof(double));
    cudaMemset(vars->gpu_aForcez, 0, atomCount * sizeof(double));
    cudaMemset(vars->gpu_mForcex, 0, molCount * sizeof(double));
    cudaMemset(vars->gpu_mForcey, 0, molCount * sizeof(double));
    cudaMemset(vars->gpu_mForcez, 0, molCount * sizeof(double));
  }

  cudaMalloc((void**) &gpu_pair1, pair1.size() * sizeof(int));
  cudaMalloc((void**) &gpu_pair2, pair2.size() * sizeof(int));
  cudaMalloc((void**) &gpu_particleCharge,
             particleCharge.size() * sizeof(double));
  cudaMalloc((void**) &gpu_particleKind, particleKind.size() * sizeof(int));
  cudaMalloc((void**) &gpu_particleMol, particleMol.size() * sizeof(int));
  cudaMalloc((void**) &gpu_REn, pair1.size() * sizeof(double));
  cudaMalloc((void**) &gpu_LJEn, pair1.size() * sizeof(double));
  cudaMalloc((void**) &gpu_final_REn, sizeof(double));
  cudaMalloc((void**) &gpu_final_LJEn, sizeof(double));

  // Copy necessary data to GPU
  cudaMemcpy(gpu_pair1, &pair1[0], pair1.size() * sizeof(int),
             cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_pair2, &pair2[0], pair2.size() * sizeof(int),
             cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleCharge, &particleCharge[0],
             particleCharge.size() * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleKind, &particleKind[0],
             particleKind.size() * sizeof(int),
             cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleMol, &particleMol[0], particleMol.size() * sizeof(int),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_x, coords.x, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, coords.y, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, coords.z, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);

  // Run the kernel...
  threadsPerBlock = 256;
  blocksPerGrid = (int)(pair1.size() / threadsPerBlock) + 1;
  BoxForceGPU <<< blocksPerGrid, threadsPerBlock>>>(gpu_pair1,
                                                    gpu_pair2,
                                                    vars->gpu_x,
                                                    vars->gpu_y,
                                                    vars->gpu_z,
                                                    boxAxes.GetAxis(box).x,
                                                    boxAxes.GetAxis(box).y,
                                                    boxAxes.GetAxis(box).z,
                                                    electrostatic,
                                                    gpu_particleCharge,
                                                    gpu_particleKind,
                                                    gpu_particleMol,
                                                    gpu_REn,
                                                    gpu_LJEn,
                                                    pair1.size(),
                                                    vars->gpu_sigmaSq,
                                                    vars->gpu_epsilon_Cn,
                                                    vars->gpu_n,
                                                    vars->gpu_VDW_Kind,
                                                    vars->gpu_isMartini,
                                                    vars->gpu_count,
                                                    vars->gpu_rCut,
                                                    vars->gpu_rCutCoulomb,
                                                    vars->gpu_rCutLow,
                                                    vars->gpu_rOn,
                                                    vars->gpu_alpha,
                                                    vars->gpu_ewald,
                                                    vars->gpu_diElectric_1,
                                                    vars->gpu_nonOrth,
                                                    vars->gpu_cell_x[box],
                                                    vars->gpu_cell_y[box],
                                                    vars->gpu_cell_z[box],
                                                    vars->gpu_Invcell_x[box],
                                                    vars->gpu_Invcell_y[box],
                                                    vars->gpu_Invcell_z[box],
                                                    vars->gpu_aForcex,
                                                    vars->gpu_aForcey,
                                                    vars->gpu_aForcez,
                                                    vars->gpu_mForcex,
                                                    vars->gpu_mForcey,
                                                    vars->gpu_mForcez,
                                                    box);


  // ReduceSum
  void * d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_REn,
                    gpu_final_REn, pair1.size());
  CubDebugExit(cudaMalloc(&d_temp_storage, temp_storage_bytes));
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_REn,
                    gpu_final_REn, pair1.size());
  cudaFree(d_temp_storage);

  // LJ ReduceSum
  d_temp_storage = NULL;
  temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_LJEn,
                    gpu_final_LJEn, pair1.size());
  CubDebugExit(cudaMalloc(&d_temp_storage, temp_storage_bytes));
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_LJEn,
                    gpu_final_LJEn, pair1.size());
  cudaFree(d_temp_storage);
  // Copy back the result to CPU ! :)
  CubDebugExit(cudaMemcpy(&cpu_final_REn, gpu_final_REn, sizeof(double),
                          cudaMemcpyDeviceToHost));
  CubDebugExit(cudaMemcpy(&cpu_final_LJEn, gpu_final_LJEn, sizeof(double),
                          cudaMemcpyDeviceToHost));
  REn = cpu_final_REn;
  LJEn = cpu_final_LJEn;

  if(copy_back) {
    CubDebugExit(cudaMemcpy(aForcex, vars->gpu_aForcex,
                            sizeof(double) * atomCount,
                            cudaMemcpyDeviceToHost));
    CubDebugExit(cudaMemcpy(aForcey, vars->gpu_aForcey,
                            sizeof(double) * atomCount,
                            cudaMemcpyDeviceToHost));
    CubDebugExit(cudaMemcpy(aForcez, vars->gpu_aForcez,
                            sizeof(double) * atomCount,
                            cudaMemcpyDeviceToHost));
    CubDebugExit(cudaMemcpy(mForcex, vars->gpu_mForcex,
                            sizeof(double) * molCount,
                            cudaMemcpyDeviceToHost));
    CubDebugExit(cudaMemcpy(mForcey, vars->gpu_mForcey,
                            sizeof(double) * molCount,
                            cudaMemcpyDeviceToHost));
    CubDebugExit(cudaMemcpy(mForcez, vars->gpu_mForcez,
                            sizeof(double) * molCount,
                            cudaMemcpyDeviceToHost));
  }
  cudaDeviceSynchronize();

  cudaFree(gpu_pair1);
  cudaFree(gpu_pair2);
  cudaFree(gpu_particleCharge);
  cudaFree(gpu_particleKind);
  cudaFree(gpu_particleMol);
  cudaFree(gpu_REn);
  cudaFree(gpu_LJEn);
  cudaFree(gpu_final_REn);
  cudaFree(gpu_final_LJEn);
}

void CallVirialReciprocalGPU(VariablesCUDA *vars,
                            XYZArray const &currentCoords,
                            XYZArray const &currentCOMDiff,
                            vector<double> &particleCharge,
                            double &rT11,
                            double &rT12,
                            double &rT13,
                            double &rT22,
                            double &rT23,
                            double &rT33,
                            uint imageSize,
                            double constVal,
                            uint box)
{
  int atomNumber = currentCoords.Count();
  int blocksPerGrid, threadsPerBlock;
  double *gpu_particleCharge;
  double *gpu_final_value;

  cudaMalloc((void**) &gpu_particleCharge,
             particleCharge.size() * sizeof(double));
  cudaMalloc((void**) &gpu_final_value, sizeof(double));

  cudaMemcpy(vars->gpu_x, currentCoords.x, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, currentCoords.y, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, currentCoords.z, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_dx, currentCOMDiff.x, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_dy, currentCOMDiff.y, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_dz, currentCOMDiff.z, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleCharge, &particleCharge[0],
             particleCharge.size() * sizeof(double),
             cudaMemcpyHostToDevice);

  // Run the kernel...
  threadsPerBlock = 256;
  blocksPerGrid = (int)(imageSize / threadsPerBlock) + 1;
  VirialReciprocalGPU <<< blocksPerGrid,
                     threadsPerBlock>>>(vars->gpu_x,
                                        vars->gpu_y,
                                        vars->gpu_z,
                                        vars->gpu_dx,
                                        vars->gpu_dy,
                                        vars->gpu_dz,
                                        vars->gpu_kxRef[box],
                                        vars->gpu_kyRef[box],
                                        vars->gpu_kzRef[box],
                                        vars->gpu_prefactRef[box],
                                        vars->gpu_hsqrRef[box],
                                        vars->gpu_sumRref[box],
                                        vars->gpu_sumIref[box],
                                        gpu_particleCharge,
                                        vars->gpu_rT11,
                                        vars->gpu_rT12,
                                        vars->gpu_rT13,
                                        vars->gpu_rT22,
                                        vars->gpu_rT23,
                                        vars->gpu_rT33,
                                        constVal,
                                        imageSize,
                                        atomNumber);

  // ReduceSum // Virial of Reciprocal
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT11,
                    gpu_final_value, imageSize);
  cudaMalloc(&d_temp_storage, temp_storage_bytes);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT11,
                    gpu_final_value, imageSize);
  cudaMemcpy(&rT11, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT12,
                    gpu_final_value, imageSize);
  cudaMemcpy(&rT12, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT13,
                    gpu_final_value, imageSize);
  cudaMemcpy(&rT13, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT22,
                    gpu_final_value, imageSize);
  cudaMemcpy(&rT22, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT23,
                    gpu_final_value, imageSize);
  cudaMemcpy(&rT23, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT33,
                    gpu_final_value, imageSize);
  cudaMemcpy(&rT33, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);

  cudaFree(gpu_particleCharge);
  cudaFree(gpu_final_value);
  cudaFree(d_temp_storage);
}

__global__ void BoxInterForceGPU(int *gpu_pair1,
                                 int *gpu_pair2,
                                 double *gpu_x,
                                 double *gpu_y,
                                 double *gpu_z,
                                 double *gpu_comx,
                                 double *gpu_comy,
                                 double *gpu_comz,
                                 double xAxes,
                                 double yAxes,
                                 double zAxes,
                                 bool electrostatic,
                                 double *gpu_particleCharge,
                                 int *gpu_particleKind,
                                 int *gpu_particleMol,
                                 double *gpu_rT11,
                                 double *gpu_rT12,
                                 double *gpu_rT13,
                                 double *gpu_rT22,
                                 double *gpu_rT23,
                                 double *gpu_rT33,
                                 double *gpu_vT11,
                                 double *gpu_vT12,
                                 double *gpu_vT13,
                                 double *gpu_vT22,
                                 double *gpu_vT23,
                                 double *gpu_vT33,
                                 int pairSize,
                                 double *gpu_sigmaSq,
                                 double *gpu_epsilon_Cn,
                                 double *gpu_n,
                                 int *gpu_VDW_Kind,
                                 int *gpu_isMartini,
                                 int *gpu_count,
                                 double *gpu_rCut,
                                 double *gpu_rCutCoulomb,
                                 double *gpu_rCutLow,
                                 double *gpu_rOn,
                                 double *gpu_alpha,
                                 int *gpu_ewald,
                                 double *gpu_diElectric_1,
                                 double *gpu_cell_x,
                                 double *gpu_cell_y,
                                 double *gpu_cell_z,
                                 double *gpu_Invcell_x,
                                 double *gpu_Invcell_y,
                                 double *gpu_Invcell_z,
                                 int *gpu_nonOrth,
                                 int box)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= pairSize)
    return;

  double distSq;
  double virX, virY, virZ;
  double pRF = 0.0, qi_qj, pVF = 0.0;
  //tensors for VDW and real part of electrostatic
  gpu_vT11[threadID] = 0.0, gpu_vT22[threadID] = 0.0, gpu_vT33[threadID] = 0.0;
  gpu_rT11[threadID] = 0.0, gpu_rT22[threadID] = 0.0, gpu_rT33[threadID] = 0.0;
  // extra tensors reserved for later on
  gpu_vT12[threadID] = 0.0, gpu_vT13[threadID] = 0.0, gpu_vT23[threadID] = 0.0;
  gpu_rT12[threadID] = 0.0, gpu_rT13[threadID] = 0.0, gpu_rT23[threadID] = 0.0;
  double diff_comx, diff_comy, diff_comz;
  double cutoff = fmax(gpu_rCut[0], gpu_rCutCoulomb[box]);

  if(InRcutGPU(distSq, virX, virY, virZ, gpu_x[gpu_pair1[threadID]],
               gpu_y[gpu_pair1[threadID]], gpu_z[gpu_pair1[threadID]],
               gpu_x[gpu_pair2[threadID]], gpu_y[gpu_pair2[threadID]],
               gpu_z[gpu_pair2[threadID]], xAxes, yAxes, zAxes, xAxes / 2.0,
               yAxes / 2.0, zAxes / 2.0, cutoff, gpu_nonOrth[0],
               gpu_cell_x, gpu_cell_y, gpu_cell_z, gpu_Invcell_x, gpu_Invcell_y,
               gpu_Invcell_z)) {
    diff_comx = gpu_comx[gpu_particleMol[gpu_pair1[threadID]]] -
                gpu_comx[gpu_particleMol[gpu_pair2[threadID]]];
    diff_comy = gpu_comy[gpu_particleMol[gpu_pair1[threadID]]] -
                gpu_comy[gpu_particleMol[gpu_pair2[threadID]]];
    diff_comz = gpu_comz[gpu_particleMol[gpu_pair1[threadID]]] -
                gpu_comz[gpu_particleMol[gpu_pair2[threadID]]];

    diff_comx = MinImageSignedGPU(diff_comx, xAxes, xAxes / 2.0);
    diff_comy = MinImageSignedGPU(diff_comy, yAxes, yAxes / 2.0);
    diff_comz = MinImageSignedGPU(diff_comz, zAxes, zAxes / 2.0);

    if(electrostatic) {
      qi_qj = gpu_particleCharge[gpu_pair1[threadID]] *
              gpu_particleCharge[gpu_pair2[threadID]];
      pRF = CalcCoulombForceGPU(distSq, qi_qj, gpu_VDW_Kind[0], gpu_ewald[0],
                                gpu_isMartini[0], gpu_alpha[box], gpu_rCutCoulomb[box],
                                gpu_diElectric_1[0]);

      gpu_rT11[threadID] = pRF * (virX * diff_comx);
      gpu_rT22[threadID] = pRF * (virY * diff_comy);
      gpu_rT33[threadID] = pRF * (virZ * diff_comz);

      //extra tensor calculations
      gpu_rT12[threadID] = pRF * (0.5 * (virX * diff_comy + virY * diff_comx));
      gpu_rT13[threadID] = pRF * (0.5 * (virX * diff_comz + virZ * diff_comx));
      gpu_rT23[threadID] = pRF * (0.5 * (virY * diff_comz + virZ * diff_comy));
    }

    pVF = CalcEnForceGPU(distSq, gpu_particleKind[gpu_pair1[threadID]],
                         gpu_particleKind[gpu_pair2[threadID]],
                         gpu_sigmaSq, gpu_n, gpu_epsilon_Cn, gpu_rCut[0],
                         gpu_rOn[0], gpu_isMartini[0], gpu_VDW_Kind[0],
                         gpu_count[0]);

    gpu_vT11[threadID] = pVF * (virX * diff_comx);
    gpu_vT22[threadID] = pVF * (virY * diff_comy);
    gpu_vT33[threadID] = pVF * (virZ * diff_comz);

    //extra tensor calculations
    gpu_vT12[threadID] = pVF * (0.5 * (virX * diff_comy + virY * diff_comx));
    gpu_vT13[threadID] = pVF * (0.5 * (virX * diff_comz + virZ * diff_comx));
    gpu_vT23[threadID] = pVF * (0.5 * (virY * diff_comz + virZ * diff_comy));
  }
}

__global__ void BoxForceGPU(int *gpu_pair1,
                            int *gpu_pair2,
                            double *gpu_x,
                            double *gpu_y,
                            double *gpu_z,
                            double xAxes,
                            double yAxes,
                            double zAxes,
                            bool electrostatic,
                            double *gpu_particleCharge,
                            int *gpu_particleKind,
                            int *gpu_particleMol,
                            double *gpu_REn,
                            double *gpu_LJEn,
                            int pairSize,
                            double *gpu_sigmaSq,
                            double *gpu_epsilon_Cn,
                            double *gpu_n,
                            int *gpu_VDW_Kind,
                            int *gpu_isMartini,
                            int *gpu_count,
                            double *gpu_rCut,
                            double *gpu_rCutCoulomb,
                            double *gpu_rCutLow,
                            double *gpu_rOn,
                            double *gpu_alpha,
                            int *gpu_ewald,
                            double *gpu_diElectric_1,
                            int *gpu_nonOrth,
                            double *gpu_cell_x,
                            double *gpu_cell_y,
                            double *gpu_cell_z,
                            double *gpu_Invcell_x,
                            double *gpu_Invcell_y,
                            double *gpu_Invcell_z,
                            double *gpu_aForcex,
                            double *gpu_aForcey,
                            double *gpu_aForcez,
                            double *gpu_mForcex,
                            double *gpu_mForcey,
                            double *gpu_mForcez,
                            int box)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= pairSize)
    return;
  double distSq;
  double qi_qj_fact;
  double qqFact = 167000.0;
  double virX = 0.0, virY = 0.0, virZ = 0.0;
  double forceRealx = 0.0, forceRealy = 0.0, forceRealz = 0.0;
  double forceLJx = 0.0, forceLJy = 0.0, forceLJz = 0.0;
  gpu_REn[threadID] = 0.0;
  gpu_LJEn[threadID] = 0.0;
  double cutoff = fmax(gpu_rCut[0], gpu_rCutCoulomb[box]);
  if(InRcutGPU(distSq, virX, virY, virZ, gpu_x[gpu_pair1[threadID]],
               gpu_y[gpu_pair1[threadID]], gpu_z[gpu_pair1[threadID]],
               gpu_x[gpu_pair2[threadID]], gpu_y[gpu_pair2[threadID]],
               gpu_z[gpu_pair2[threadID]], xAxes, yAxes, zAxes, xAxes / 2.0,
               yAxes / 2.0, zAxes / 2.0, cutoff, gpu_nonOrth[0], gpu_cell_x,
               gpu_cell_y, gpu_cell_z, gpu_Invcell_x, gpu_Invcell_y,
               gpu_Invcell_z)) {
    if(electrostatic) {
      qi_qj_fact = gpu_particleCharge[gpu_pair1[threadID]] *
                   gpu_particleCharge[gpu_pair2[threadID]] * qqFact;
      gpu_REn[threadID] = CalcCoulombGPU(distSq, qi_qj_fact, gpu_rCutLow[0],
                                         gpu_ewald[0], gpu_VDW_Kind[0],
                                         gpu_alpha[box],
                                         gpu_rCutCoulomb[box],
                                         gpu_isMartini[0],
                                         gpu_diElectric_1[0]);
    }
    gpu_LJEn[threadID] = CalcEnGPU(distSq,
                                   gpu_particleKind[gpu_pair1[threadID]],
                                   gpu_particleKind[gpu_pair2[threadID]],
                                   gpu_sigmaSq, gpu_n, gpu_epsilon_Cn,
                                   gpu_VDW_Kind[0], gpu_isMartini[0],
                                   gpu_rCut[0], gpu_rOn[0], gpu_count[0]);
    if(electrostatic) {
      double coulombVir = CalcCoulombForceGPU(distSq, qi_qj_fact,
                                              gpu_VDW_Kind[0], gpu_ewald[0],
                                              gpu_isMartini[0],
                                              gpu_alpha[box],
                                              gpu_rCutCoulomb[box],
                                              gpu_diElectric_1[0]);
      forceRealx = virX * coulombVir;
      forceRealy = virY * coulombVir;
      forceRealz = virZ * coulombVir;
    }
    double pVF = CalcEnForceGPU(distSq, gpu_particleKind[gpu_pair1[threadID]],
                                gpu_particleKind[gpu_pair2[threadID]],
                                gpu_sigmaSq, gpu_n, gpu_epsilon_Cn,
                                gpu_rCut[0], gpu_rOn[0], gpu_isMartini[0],
                                gpu_VDW_Kind[0], gpu_count[0]);
    forceLJx = virX * pVF;
    forceLJy = virY * pVF;
    forceLJz = virZ * pVF;

    atomicAdd(&gpu_aForcex[gpu_pair1[threadID]], forceRealx + forceLJx);
    atomicAdd(&gpu_aForcey[gpu_pair1[threadID]], forceRealy + forceLJy);
    atomicAdd(&gpu_aForcez[gpu_pair1[threadID]], forceRealz + forceLJz);
    atomicAdd(&gpu_aForcex[gpu_pair2[threadID]],-1.0*(forceRealx + forceLJx));
    atomicAdd(&gpu_aForcey[gpu_pair2[threadID]],-1.0*(forceRealy + forceLJy));
    atomicAdd(&gpu_aForcez[gpu_pair2[threadID]],-1.0*(forceRealz + forceLJz));

    atomicAdd(&gpu_mForcex[gpu_particleMol[gpu_pair1[threadID]]],
              forceRealx + forceLJx);
    atomicAdd(&gpu_mForcey[gpu_particleMol[gpu_pair1[threadID]]],
              forceRealy + forceLJy);
    atomicAdd(&gpu_mForcez[gpu_particleMol[gpu_pair1[threadID]]],
              forceRealz + forceLJz);
    atomicAdd(&gpu_mForcex[gpu_particleMol[gpu_pair2[threadID]]],
              -1.0 * (forceRealx + forceLJx));
    atomicAdd(&gpu_mForcey[gpu_particleMol[gpu_pair2[threadID]]],
              -1.0 * (forceRealy + forceLJy));
    atomicAdd(&gpu_mForcez[gpu_particleMol[gpu_pair2[threadID]]],
              -1.0 * (forceRealz + forceLJz));
  }
}

__global__ void VirialReciprocalGPU(double *gpu_x,
                                   double *gpu_y,
                                   double *gpu_z,
                                   double *gpu_comDx,
                                   double *gpu_comDy,
                                   double *gpu_comDz,
                                   double *gpu_kxRef,
                                   double *gpu_kyRef,
                                   double *gpu_kzRef,
                                   double *gpu_prefactRef,
                                   double *gpu_hsqrRef,
                                   double *gpu_sumRref,
                                   double *gpu_sumIref,
                                   double *gpu_particleCharge,
                                   double *gpu_rT11,
                                   double *gpu_rT12,
                                   double *gpu_rT13,
                                   double *gpu_rT22,
                                   double *gpu_rT23,
                                   double *gpu_rT33,
                                   double constVal,
                                   uint imageSize,
                                   uint atomNumber)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= imageSize)
    return;

  double factor, arg;
  int i;
  factor = gpu_prefactRef[threadID] * (gpu_sumRref[threadID] *
                                       gpu_sumRref[threadID] +
                                       gpu_sumIref[threadID] *
                                       gpu_sumIref[threadID]);
  gpu_rT11[threadID] = factor * (1.0 - 2.0 *
                                 (constVal + 1.0 / gpu_hsqrRef[threadID]) *
                                 gpu_kxRef[threadID] * gpu_kxRef[threadID]);
  gpu_rT12[threadID] = factor * (-2.0 *
                                 (constVal + 1.0 / gpu_hsqrRef[threadID]) *
                                 gpu_kxRef[threadID] * gpu_kyRef[threadID]);
  gpu_rT13[threadID] = factor * (-2.0 *
                                 (constVal + 1.0 / gpu_hsqrRef[threadID]) *
                                 gpu_kxRef[threadID] * gpu_kzRef[threadID]);
  gpu_rT22[threadID] = factor * (1.0 - 2.0 *
                                 (constVal + 1.0 / gpu_hsqrRef[threadID]) *
                                 gpu_kyRef[threadID] * gpu_kyRef[threadID]);
  gpu_rT23[threadID] = factor * (-2.0 *
                                 (constVal + 1.0 / gpu_hsqrRef[threadID]) *
                                 gpu_kyRef[threadID] * gpu_kzRef[threadID]);
  gpu_rT33[threadID] = factor * (1.0 - 2.0 *
                                 (constVal + 1.0 / gpu_hsqrRef[threadID]) *
                                 gpu_kzRef[threadID] * gpu_kzRef[threadID]);

  //Intramolecular part
  for(i = 0; i < atomNumber; i++) {
    arg = DotProductGPU(gpu_kxRef[threadID], gpu_kyRef[threadID],
                        gpu_kzRef[threadID], gpu_x[i], gpu_y[i], gpu_z[i]);

    factor = gpu_prefactRef[threadID] * 2.0 *
             (gpu_sumIref[threadID] * cos(arg) - gpu_sumRref[threadID] * sin(arg)) *
             gpu_particleCharge[i];

    gpu_rT11[threadID] += factor * (gpu_kxRef[threadID] * gpu_comDx[i]);
    gpu_rT12[threadID] += factor * 0.5 * (gpu_kxRef[threadID] * gpu_comDy[i] +
                                          gpu_kyRef[threadID] * gpu_comDx[i]);
    gpu_rT13[threadID] += factor * 0.5 * (gpu_kxRef[threadID] * gpu_comDz[i] +
                                          gpu_kzRef[threadID] * gpu_comDx[i]);
    gpu_rT22[threadID] += factor * (gpu_kyRef[threadID] * gpu_comDy[i]);
    gpu_rT13[threadID] += factor * 0.5 * (gpu_kyRef[threadID] * gpu_comDz[i] +
                                          gpu_kzRef[threadID] * gpu_comDy[i]);
    gpu_rT33[threadID] += factor * (gpu_kzRef[threadID] * gpu_comDz[i]);
  }
}

__device__ double CalcEnForceGPU(double distSq, int kind1, int kind2,
                                 double *gpu_sigmaSq, double *gpu_n,
                                 double *gpu_epsilon_Cn, double gpu_rCut,
                                 double gpu_rOn, int gpu_isMartini,
                                 int gpu_VDW_Kind, int gpu_count)
{
  if((gpu_rCut * gpu_rCut) < distSq) {
    return 0.0;
  }

  int index = FlatIndexGPU(kind1, kind2, gpu_count);
  if(gpu_VDW_Kind == GPU_VDW_STD_KIND) {
    return CalcVirParticleGPU(distSq, index, gpu_sigmaSq, gpu_n,
                              gpu_epsilon_Cn);
  } else if(gpu_VDW_Kind == GPU_VDW_SHIFT_KIND) {
    return CalcVirShiftGPU(distSq, index, gpu_sigmaSq, gpu_n, gpu_epsilon_Cn);
  } else if(gpu_VDW_Kind == GPU_VDW_SWITCH_KIND && gpu_isMartini) {
    return CalcVirSwitchMartiniGPU(distSq, index, gpu_sigmaSq, gpu_n,
                                   gpu_epsilon_Cn, gpu_rCut, gpu_rOn);
  } else
    return CalcVirSwitchGPU(distSq, index, gpu_sigmaSq, gpu_epsilon_Cn, gpu_n,
                            gpu_rCut, gpu_rOn);
}

//ElectroStatic Calculation
//**************************************************************//
__device__ double CalcCoulombVirParticleGPU(double distSq, double qi_qj,
    double gpu_alpha)
{
  double dist = sqrt(distSq);
  double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
  double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
  double temp = 1.0 - erf(gpu_alpha * dist);
  return qi_qj * (temp / dist + constValue * expConstValue) / distSq;
}

__device__ double CalcCoulombVirShiftGPU(double distSq, double qi_qj,
    int gpu_ewald, double gpu_alpha)
{
  if(gpu_ewald) {
    double dist = sqrt(distSq);
    double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
    double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
    double temp = 1.0 - erf(gpu_alpha * dist);
    return qi_qj * (temp / dist + constValue * expConstValue) / distSq;
  } else {
    double dist = sqrt(distSq);
    return qi_qj / (distSq * dist);
  }
}
__device__ double CalcCoulombVirSwitchMartiniGPU(double distSq, double qi_qj,
    int gpu_ewald,
    double gpu_alpha,
    double gpu_rCut,
    double gpu_diElectric_1)
{
  if(gpu_ewald) {
    double dist = sqrt(distSq);
    double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
    double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
    double temp = 1.0 - erf(gpu_alpha * dist);
    return  qi_qj * (temp / dist + constValue * expConstValue) / distSq;
  } else {
    // in Martini, the Coulomb switching distance is zero, so we will have
    // sqrt(distSq) - rOnCoul =  sqrt(distSq)
    double dist = sqrt(distSq);
    double rij_ronCoul_2 = distSq;
    double rij_ronCoul_3 = dist * distSq;

    double A1 = 1.0 * (-(1.0 + 4) * gpu_rCut) / (pow(gpu_rCut, 1.0 + 2) *
                pow(gpu_rCut, 2));
    double B1 = -1.0 * (-(1.0 + 3) * gpu_rCut) / (pow(gpu_rCut, 1.0 + 2) *
                pow(gpu_rCut, 3));

    double virCoul = A1 / rij_ronCoul_2 + B1 / rij_ronCoul_3;
    return qi_qj * gpu_diElectric_1 * ( 1.0 / (dist * distSq) + virCoul / dist);
  }
}

__device__ double CalcCoulombVirSwitchGPU(double distSq, double qi_qj,
                                          int gpu_ewald, double gpu_alpha,
                                          double gpu_rCut)
{
  if(gpu_ewald) {
    double dist = sqrt(distSq);
    double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
    double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
    double temp = 1.0 - erf(gpu_alpha * dist);
    return qi_qj * (temp / dist + constValue * expConstValue) / distSq;
  } else {
    double rCutSq = gpu_rCut * gpu_rCut;
    double dist = sqrt(distSq);
    double switchVal = distSq / rCutSq - 1.0;
    switchVal *= switchVal;

    double dSwitchVal = 2.0 * (distSq / rCutSq - 1.0) * 2.0 * dist / rCutSq;
    return -1.0 * qi_qj * (dSwitchVal / distSq - switchVal / (distSq * dist));
  }
}

//VDW Calculation
//*****************************************************************//
__device__ double CalcVirParticleGPU(double distSq, int index,
                                     double *gpu_sigmaSq, double *gpu_n,
                                     double *gpu_epsilon_Cn)
{
  double rNeg2 = 1.0 / distSq;
  double rRat2 = gpu_sigmaSq[index] * rNeg2;
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;
  double repulse = pow(rRat2, gpu_n[index] / 2.0);
  return gpu_epsilon_Cn[index] * 6.0 *
         ((gpu_n[index] / 6.0) * repulse - attract) * rNeg2;
}

__device__ double CalcVirShiftGPU(double distSq, int index, double *gpu_sigmaSq,
                                  double *gpu_n, double *gpu_epsilon_Cn)
{
  double rNeg2 = 1.0 / distSq;
  double rRat2 = gpu_sigmaSq[index] * rNeg2;
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;
  double repulse = pow(rRat2, gpu_n[index] / 2.0);
  return gpu_epsilon_Cn[index] * 6.0 *
         ((gpu_n[index] / 6.0) * repulse - attract) * rNeg2;
}

__device__ double CalcVirSwitchMartiniGPU(double distSq, int index,
    double *gpu_sigmaSq, double *gpu_n,
    double *gpu_epsilon_Cn,
    double gpu_rCut, double gpu_rOn)
{
  double r_1 = 1.0 / sqrt(distSq);
  double r_8 = pow(r_1, 8);
  double r_n2 = pow(r_1, gpu_n[index] + 2);

  double rij_ron = sqrt(distSq) - gpu_rOn;
  double rij_ron_2 = rij_ron * rij_ron;
  double rij_ron_3 = rij_ron_2 * rij_ron;

  double pn = gpu_n[index];
  double An = pn * ((pn + 1) * gpu_rOn - (pn + 4) * gpu_rCut) /
              (pow(gpu_rCut, pn + 2) * pow(gpu_rCut - gpu_rOn, 2));
  double Bn = -pn * ((pn + 1) * gpu_rOn - (pn + 3) * gpu_rCut) /
              (pow(gpu_rCut, pn + 2) * pow(gpu_rCut - gpu_rOn, 3));

  double sig6 = pow(gpu_sigmaSq[index], 3);
  double sign = pow(gpu_sigmaSq[index], pn / 2);

  double A6 = 6.0 * ((6.0 + 1) * gpu_rOn - (6.0 + 4) * gpu_rCut) /
              (pow(gpu_rCut, 6.0 + 2) * pow(gpu_rCut - gpu_rOn, 2));
  double B6 = -6.0 * ((6.0 + 1) * gpu_rOn - (6.0 + 3) * gpu_rCut) /
              (pow(gpu_rCut, 6.0 + 2) * pow(gpu_rCut - gpu_rOn, 3));

  double dshifttempRep = An * rij_ron_2 + Bn * rij_ron_3;
  double dshifttempAtt = A6 * rij_ron_2 + B6 * rij_ron_3;

  const double dshiftRep = ( distSq > gpu_rOn * gpu_rOn ?
                             dshifttempRep * r_1 : 0);
  const double dshiftAtt = ( distSq > gpu_rOn * gpu_rOn ?
                             dshifttempAtt * r_1 : 0);
  double Wij = gpu_epsilon_Cn[index] * (sign * (pn * r_n2 + dshiftRep) -
                                        sig6 * (6.0 * r_8 + dshiftAtt));
  return Wij;
}

__device__ double CalcVirSwitchGPU(double distSq, int index,
                                   double *gpu_sigmaSq, double *gpu_epsilon_Cn,
                                   double *gpu_n, double gpu_rCut,
                                   double gpu_rOn)
{
  double rCutSq = gpu_rCut * gpu_rCut;
  double rCutSq_rijSq = rCutSq - distSq;
  double rCutSq_rijSq_Sq = rCutSq_rijSq * rCutSq_rijSq;
  double rOnSq = gpu_rOn * gpu_rOn;

  double rNeg2 = 1.0 / distSq;
  double rRat2 = rNeg2 * gpu_sigmaSq[index];
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;
  double repulse = pow(rRat2, gpu_n[index] / 2.0);
  double factor1 = rCutSq - 3 * rOnSq;
  double factor2 = pow((rCutSq - rOnSq), -3);

  double fE = rCutSq_rijSq_Sq * factor2 * (factor1 + 2 * distSq);
  double fW = 12.0 * factor2 * rCutSq_rijSq * (rOnSq - distSq);

  const double factE = ( distSq > rOnSq ? fE : 1.0);
  const double factW = ( distSq > rOnSq ? fW : 0.0);

  double Wij = gpu_epsilon_Cn[index] * 6.0 *
               ((gpu_n[index] / 6.0) * repulse - attract) * rNeg2;
  double Eij = gpu_epsilon_Cn[index] * (repulse - attract);

  return (Wij * factE - Eij * factW);
}

#endif
